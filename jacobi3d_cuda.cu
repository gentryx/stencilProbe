
#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>
#include <vector>

#define GET(X, Y, Z) gridOld[(X) + (Y) * dimX + (Z) * dimX * dimY]
#define SET(X, Y, Z) gridNew[(X) + (Y) * dimX + (Z) * dimX * dimY]

__global__ void update(double *gridOld, double *gridNew, int dimX, int dimY, int dimZ)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    if ((x == 0) || (x >= (dimX - 1)) ||
        (y == 0) || (y >= (dimY - 1)) ||
        (z == 0) || (z >= (dimZ - 1))) {
        return;
    }

    SET(x, y, z) = (GET(x, y, z - 1) +
                    GET(x, y - 1, z) +
                    GET(x - 1, y, z) +
                    GET(x + 1, y, z) +
                    GET(x, y + 1, z) +
                    GET(x, y, z + 1)) * (1.0 / 6.0);
}

void init(double *gridNew, int dimX, int dimY, int dimZ)
{
    for (int z = 0; z < dimZ; ++z) {
        for (int y = 0; y < dimY; ++y) {
            for (int x = 0; x < dimX; ++x) {
                double value = 0;
                if ((x * y * z) == 0) {
                    value = 1;
                }
                SET(x, y, z) = value;
            }
        }
    }
}

void print(double *gridOld, int dimX, int dimY, int dimZ)
{
    for (int z = 0; z < dimZ; ++z) {
        for (int y = 0; y < dimY; ++y) {
            for (int x = 0; x < dimX; ++x) {
                std::cout << " " << GET(x, y, z);
            }
            std::cout << "\n";
        }
    }
}

int divAndRoundUp(int dim, int blockDim)
{
    int res = dim / blockDim;
    if (dim % blockDim) {
        res += 1;
    }
    return res;
}

void benchmark(std::vector<double> *gridOld, std::vector<double> *gridNew, int dimX, int dimY, int dimZ, int repeats)
{
    int byteSize = dimX * dimY * dimZ * sizeof(double);
    double *devGridOld;
    double *devGridNew;
    hipMalloc(&devGridOld, byteSize);
    hipMalloc(&devGridNew, byteSize);
    hipMemcpy(devGridOld, &gridOld->front(), byteSize, hipMemcpyHostToDevice);
    hipMemcpy(devGridNew, &gridNew->front(), byteSize, hipMemcpyHostToDevice);
    dim3 blockDim(32, 32, 1);
    dim3 gridDim(divAndRoundUp(dimX, blockDim.x),
                 divAndRoundUp(dimY, blockDim.y),
                 divAndRoundUp(dimZ, blockDim.z));

    for (int t = 0; t < repeats; ++t) {
        update<<<gridDim, blockDim>>>(devGridOld, devGridNew, dimX, dimY, dimZ);
        std::swap(devGridOld, devGridNew);
    }

    hipMemcpy(&gridOld->front(), devGridOld, byteSize, hipMemcpyDeviceToHost);
    hipFree(devGridOld);
    hipFree(devGridNew);
}

int main(int argc, char **argv)
{
    if (argc != 5) {
        std::cerr << "usage: " << argv[0] << " DIM_X DIM_Y DIM_Z REPEATS\n";
        return 1;
    }
    std::stringstream buf;
    for (int i = 1; i <= 4; ++i)
        buf << argv[i] << " ";
    int dimX, dimY, dimZ, repeats;
    buf >> dimX;
    buf >> dimY;
    buf >> dimZ;
    buf >> repeats;

    int size = dimX * dimY * dimZ;
    std::vector<double> gridOld(size);
    std::vector<double> gridNew(size);
    init(&gridOld[0], dimX, dimY, dimZ);
    init(&gridNew[0], dimX, dimY, dimZ);

    benchmark(&gridOld, &gridNew, dimX, dimY, dimZ, repeats);

    print(&gridOld[0], dimX, dimY, dimZ);
}