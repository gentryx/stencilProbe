#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <stdexcept>
#include <vector>

#include "eval.h"

#define GET(X, Y, Z) gridOld[((X + dimX) % dimX) + ((Y + dimY) % dimY) * dimX + ((Z + dimZ) % dimZ) * dimX * dimY]
#define SET(X, Y, Z) gridNew[((X + dimX) % dimX) + ((Y + dimY) % dimY) * dimX + ((Z + dimZ) % dimZ) * dimX * dimY]

__global__ void update(double *gridOld, double *gridNew, int dimX, int dimY, int dimZ, int wavefrontLength)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    int zStart = blockIdx.z * wavefrontLength;
    int zEnd = zStart + wavefrontLength;
    if (zEnd > dimZ) {
        zEnd = dimZ;
    }

    for (int z = zStart; z < zEnd; ++z) {
        SET(x, y, z) = (GET(x, y, z - 1) +
                        GET(x, y - 1, z) +
                        GET(x - 1, y, z) +
                        GET(x + 1, y, z) +
                        GET(x, y + 1, z) +
                        GET(x, y, z + 1)) * (1.0 / 6.0);
    }
}

void init(double *gridNew, int dimX, int dimY, int dimZ)
{
    for (int z = 0; z < dimZ; ++z) {
        for (int y = 0; y < dimY; ++y) {
            for (int x = 0; x < dimX; ++x) {
                double value = 0;
                if ((x * y * z) == 0) {
                    value = 1;
                }
                SET(x, y, z) = value;
            }
        }
    }
}

void print(double *gridOld, int dimX, int dimY, int dimZ)
{
    for (int z = 0; z < dimZ; ++z) {
        for (int y = 0; y < dimY; ++y) {
            for (int x = 0; x < dimX; ++x) {
                std::cout << " " << GET(x, y, z);
            }
            std::cout << "\n";
        }
    }
}

int divAndRoundUp(int dim, int blockDim)
{
    int res = dim / blockDim;
    if (dim % blockDim) {
        res += 1;
    }
    return res;
}

void checkForCUDAError()
{
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "ERROR: " << hipGetErrorString(error) << "\n";
        throw std::runtime_error("CUDA error");
    }
}

void benchmark(std::vector<double> *gridOld, std::vector<double> *gridNew, int dimX, int dimY, int dimZ, int repeats, dim3 blockDim, int wavefrontLength)
{
    checkForCUDAError();
    hipDeviceSynchronize();
    double tStartInit = getUTtime();

    int byteSize = dimX * dimY * dimZ * sizeof(double);
    double *devGridOld;
    double *devGridNew;
    hipMalloc(&devGridOld, byteSize);
    hipMalloc(&devGridNew, byteSize);
    hipMemcpy(devGridOld, &gridOld->front(), byteSize, hipMemcpyHostToDevice);
    hipMemcpy(devGridNew, &gridNew->front(), byteSize, hipMemcpyHostToDevice);
    dim3 gridDim(divAndRoundUp(dimX, blockDim.x),
                 divAndRoundUp(dimY, blockDim.y),
                 divAndRoundUp(dimZ, blockDim.z * wavefrontLength));

    hipDeviceSynchronize();
    double tStartCalc = getUTtime();

    for (int t = 0; t < repeats; ++t) {
        update<<<gridDim, blockDim>>>(devGridOld, devGridNew, dimX, dimY, dimZ, wavefrontLength);
        std::swap(devGridOld, devGridNew);
    }

    hipDeviceSynchronize();
    double tEndCalc = getUTtime();

    hipMemcpy(&gridOld->front(), devGridOld, byteSize, hipMemcpyDeviceToHost);
    hipFree(devGridOld);
    hipFree(devGridNew);

    hipDeviceSynchronize();
    double tEnd = getUTtime();
    checkForCUDAError();
    eval(tStartInit, tStartCalc, tEndCalc, tEnd, dimX, dimY, dimZ, repeats);
}

int main(int argc, char **argv)
{
    if ((argc < 6) || (argc > 9)) {
        std::cerr << "usage: " << argv[0] << " DIM_X DIM_Y DIM_Z REPEATS CUDA_DEVICE [BLOCK_DIM_X=32] [BLOCK_DIM_Y=32] [WAVEFRONT_LENGTH=1] \n";
        return 1;
    }
    std::stringstream buf;
    for (int i = 1; i < argc; ++i) {
        buf << argv[i] << " ";
    }
    int dimX, dimY, dimZ, repeats, cudaDevice, wavefrontLength;
    buf >> dimX;
    buf >> dimY;
    buf >> dimZ;
    buf >> repeats;
    buf >> cudaDevice;
    hipSetDevice(cudaDevice);
    dim3 blockDim(32, 32, 1);

    if (argc > 5) {
        buf >> blockDim.x;
    }

    if (argc > 6) {
        buf >> blockDim.y;
    }

    if (argc > 7) {
        buf >> wavefrontLength;
    }

    int size = dimX * dimY * dimZ;
    std::vector<double> gridOld(size);
    std::vector<double> gridNew(size);
    init(&gridOld[0], dimX, dimY, dimZ);
    init(&gridNew[0], dimX, dimY, dimZ);

    benchmark(&gridOld, &gridNew, dimX, dimY, dimZ, repeats, blockDim, wavefrontLength);

    print(&gridOld[0], dimX, dimY, dimZ);
}
